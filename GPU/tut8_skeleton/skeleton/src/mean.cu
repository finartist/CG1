#include "hip/hip_runtime.h"
/// @file
////////////////////////////////////////////////////////////////////////////////////////////////////
///
/// Copyright (C) 2016/17      Christian Lessig, Otto-von-Guericke Universitaet Magdeburg
///
////////////////////////////////////////////////////////////////////////////////////////////////////
///
///  module     : lecture 8
///
///  author     : lessig@isg.cs.ovgu.de
///
///  project    : GPU Programming
///
///  description: CUDA basics
///
////////////////////////////////////////////////////////////////////////////////////////////////////

#include <iostream>
#include <algorithm>

#include "cuda_util.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////////////////////////
//! Non-local means filter in 1D
////////////////////////////////////////////////////////////////////////////////////////////////////
template< int ApronSize >
void
meanRef( float* data, float* means, int n) {

  for( int i = 0; i < n; ++i) {
    means[i] = 0.0;
    for( int k = i-ApronSize; k <= i+ApronSize; ++k) {
      if( k >= 0 && k < n) {
        means[i] += data[k];
      }
    }
    means[i] /= 2*ApronSize+1;
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//! Non-local means filter in 1D
////////////////////////////////////////////////////////////////////////////////////////////////////
template< int ApronSize > //use this as template function, so compiler can optimize the loop
__global__
void
mean( float* gdata, int n) {

	__shared__ float smem[1024 + 2*ApronSize];

	//have to read from global memory -> use global ID, but write to block specific memory -> use local id
	int gtid = blockDim.x * blockIdx.x + threadIdx.x;
	smem[threadIdx.x + ApronSize] = gdata[gtid];
	
	//padding
	 // very first block
	if(blockIdx.x == 0)
	{
		if (threadIdx.x < ApronSize)
		{
			smem[threadIdx.x] = 0.0f;
		}
	}
	 //every other block
	else
	{
		
	}

	 //very last block
	if(blockIdx.x == gridDim.x-1)
	{
		if (threadIdx.x >= n - ApronSize)
		{
			smem[n + ApronSize + (n - threadIdx.x - 1)] = 0.0f;
		}
	}
	else
	{
		
	}

	//if more threads than one warp (32) -> need to sync them to avoid errors
	__syncthreads();

	float meanval = 0.0f;
	int start = threadIdx.x - ApronSize;
	int end = threadIdx.x + ApronSize + 1;
	for(int j = start; j < end; j++)
	{
		meanval += smem[j + ApronSize];
	}
	meanval /= static_cast<float> (2 * ApronSize + 1);
	gdata[threadIdx.x] = meanval;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// program entry point
////////////////////////////////////////////////////////////////////////////////////////////////////
int
main( int /*argc*/, char** /*argv*/ ) {

  const int n = 128;
  float* data = (float*) malloc( sizeof(float) * n);
  for( unsigned int i = 0; i < n; ++i) {
    data[i] = (float) i;
  }

  // compute reference solution
  float* meansref = (float*) malloc( sizeof(float) * n);
  meanRef<2>( data, meansref, n);

  // check execution environment
  // get number of CUDA devices
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if(deviceCount < 1)
  {
	  exit(0);
  }

  // query the device properties
  //hipDeviceProp_t props;
  //hipGetDeviceProperties(&props, 0);
  //printDeviceProps(props);

  // set the device
  hipSetDevice(0);

  // allocate device memory
  float* device_data = nullptr;
  hipMalloc((void**) &device_data, n * sizeof(float));

  // copy device memory
  hipMemcpy(device_data, data, n * sizeof(float), hipMemcpyHostToDevice);

  // determine thread layout
  const int max_threads_per_block = 1024; //should better be read from device properties
  int num_blocks = n / max_threads_per_block;
  if(n % max_threads_per_block != 0)
  {
	  num_blocks++;
  }
  int num_threads_per_block = std::min(n, max_threads_per_block);

  // run kernel
  // mean is a template function, so need to put the template argument there
  // third argument would be size if shared memory
  const int apron_size = 2;
  mean<2><<<num_blocks, num_threads_per_block, n + 2*apron_size >>>(device_data, n);
  checkLastCudaError("kernel execution failed");

  // copy result back to host
  hipMemcpy(data, device_data, n * sizeof(float), hipMemcpyDeviceToHost);

  // output
  bool correct = true;
  for( unsigned int i = 0; i < n; ++i) {
    correct &= (data[i] == meansref[i]);
  }
  std::cout << (correct ? "correct" : "incorrect") << std::endl;

  // clean up device memory
  hipFree(device_data);

  // clean up host memory
  free( meansref);
  free( data);

  std::getchar();

  return EXIT_SUCCESS;
}
