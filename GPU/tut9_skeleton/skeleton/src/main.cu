#include "hip/hip_runtime.h"
/// @file
////////////////////////////////////////////////////////////////////////////////////////////////////
///
/// Copyright (C) 2017/18      Christian Lessig, Otto-von-Guericke Universitaet Magdeburg
///
////////////////////////////////////////////////////////////////////////////////////////////////////
///
///  module     : tutorial 9
///
///  author     : lessig@isg.cs.ovgu.de
///
///  project    : GPU Programming
///
///  description: CUDA convolution
///
////////////////////////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <iostream>
#include <algorithm>
#include <chrono>

typedef std::chrono::time_point<std::chrono::high_resolution_clock> tpoint;

// includes, project
#include "cuda_util.h"
#include "kernel_separable.h"
#include "image.h"

// host implementation
extern void
convSeparableHost( float* kdata, const int& kernel_supp, const Image& image, Image& image_conv);

////////////////////////////////////////////////////////////////////////////////////////////////////
// convolution
////////////////////////////////////////////////////////////////////////////////////////////////////
//improve performance with setting "Apron size" == kernel_supp_half as template parameter
template<int kernel_supp_half>
__global__
void
convSeparable( float* kernel,
               float* image, float* image_conv, const unsigned int image_size) {

	int pixel_x = blockDim.x*blockIdx.x + threadIdx.x;
	int pixel_y = blockDim.y*blockIdx.y + threadIdx.y;

	__shared__ float weights[2*kernel_supp_half +1];
	if(threadIdx.x < 2*kernel_supp_half+1)
	{
		weights[threadIdx.x] = kernel[threadIdx.x];
	}

	// reinterpret weights as kernel (only works if memory layout is the same)
	// useful to have code available without rewriting it every time
	KernelSeperable<kernel_supp_half>* kernelshared = (KernelSeperable<kernel_supp_half> *) &weights;
	kernelshared->apply(pixel_x, pixel_y, image, image_conv, image_size);

#if 0
	int ikx = 0;
	int iky = 0;
	float val = 0.0;
	//kernelsupp_half = apron size
    for(int iy = pixel_y - kernel_supp_half; iy <= kernel_supp_half + pixel_y; ++iy, iky++)
    {
		float wy = kernel[iky];
	    for(int ix = pixel_x -kernel_supp_half; ix <= kernel_supp_half + pixel_x; ++ix, ikx++)
	    {
			if(ix >= 0 && ix < image_size && iy >= 0 && iy < image_size)
			{
				float wx = weights[ikx];
				val += wx * wy * image[pixel_y * image_size + pixel_x];
			}
	    }
		ikx = 0;
    }
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// initialize Cuda device
////////////////////////////////////////////////////////////////////////////////////////////////////
bool
initDevice( int& device_handle, int& max_threads_per_block) {

  int deviceCount = 0;
  checkErrorsCuda( hipGetDeviceCount(&deviceCount));

  if( 0 == deviceCount) {
    std::cerr << "initDevice() : No CUDA device found." << std::endl;
    return false;
  }

  // one could implement more complex logic here to find the fastest device
  if( deviceCount > 1) {
    std::cerr << "initDevice() : Multiple CUDA devices found. Using first one." << std::endl;
  }

  // set the device
  checkErrorsCuda( hipSetDevice( device_handle));

  hipDeviceProp_t device_props;
  checkErrorsCuda( hipGetDeviceProperties(&device_props, device_handle));
  max_threads_per_block = device_props.maxThreadsPerBlock;

  return true;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// program entry point
////////////////////////////////////////////////////////////////////////////////////////////////////
int
main( int /*argc*/, char** /*argv*/ ) {

  const int kernel_supp = 5;
  const int kernel_supp_half = kernel_supp / 2;
  float kdata[] = {0.0103339f, 0.207561f, 0.56421f, 0.207561f, 0.0103339f};

  Image image( "../images/im.pgm");

  Image image_conv( image.n_rows, image.n_cols);
  convSeparableHost( kdata, kernel_supp_half, image, image_conv);
  image_conv.write( "../images/im_conv_host.pgm");


  // check execution environment
  int device_handle = 0;
  int max_threads_per_block = 0;
  if( ! initDevice( device_handle, max_threads_per_block)) {
    return EXIT_FAILURE;
  }

  // initialize memory
  float* kernel_device = nullptr;
  float* image_device = nullptr;
  float* image_conv_device = nullptr;

  // allocate device memory
  checkErrorsCuda( hipMalloc((void **) &kernel_device, sizeof(float) * kernel_supp));
  checkErrorsCuda( hipMalloc((void **) &image_device, sizeof(float) * image.n_cols * image.n_rows));
  checkErrorsCuda( hipMalloc((void **) &image_conv_device, sizeof(float) * image.n_cols * image.n_rows));

  // copy device memory
  checkErrorsCuda( hipMemcpy( (void*) kernel_device, kdata,
                                sizeof(float) * kernel_supp,
                                hipMemcpyHostToDevice ));
  checkErrorsCuda( hipMemcpy( (void*) image_device, &(image.data[0]),
                                sizeof(float) * image.n_cols * image.n_rows,
                                hipMemcpyHostToDevice ));

  // determine thread layout
  max_threads_per_block = 1024;
  const int max_threads_per_block_sqrt = 32;

  dim3 num_threads_per_block( 1, 1, 1);
  dim3 num_blocks( 1, 1, 1);
  
  //cut image in 32x32 blocks
  num_threads_per_block.x = max_threads_per_block_sqrt;
  num_threads_per_block.y = max_threads_per_block_sqrt;
  num_blocks.x = image.n_cols / max_threads_per_block_sqrt;
  num_blocks.y = image.n_rows / max_threads_per_block_sqrt;

  // run kernel
  assert( image.n_rows == image.n_cols);
  tpoint t_start = std::chrono::high_resolution_clock::now();
  convSeparable<kernel_supp_half><<< num_blocks , num_threads_per_block >>>( kernel_device, /*kernel_supp_half,*/
                                                           image_device, image_conv_device, image.n_rows);

  tpoint t_end = std::chrono::high_resolution_clock::now();
  double wall_clock = std::chrono::duration<double, std::milli>(t_end-t_start).count();
  std::cerr << "Execution time: " <<  wall_clock << " ms."<< std::endl;

  checkLastCudaError("Kernel execution failed");
  hipDeviceSynchronize();

  // copy result back to host
  checkErrorsCuda( hipMemcpy( &image_conv.data[0], image_conv_device,
                               sizeof(float) * image.n_cols * image.n_rows,
                               hipMemcpyDeviceToHost ));
  // write result
  image_conv.write( "../images/im_conv_device.pgm");

  // clean up device memory
  checkErrorsCuda( hipFree( kernel_device));
  checkErrorsCuda( hipFree( image_device));
  checkErrorsCuda( hipFree( image_conv_device));

  return EXIT_SUCCESS;
}
