#include "hip/hip_runtime.h"
/// @file
////////////////////////////////////////////////////////////////////////////////////////////////////
///
/// Copyright (C) 2017/18      Christian Lessig, Otto-von-Guericke Universitaet Magdeburg
///
////////////////////////////////////////////////////////////////////////////////////////////////////
///
///  module     : tutorial 7
///
///  author     : lessig@isg.cs.ovgu.de
///
///  project    : GPU Programming
///
///  description: CUDA basics
///
////////////////////////////////////////////////////////////////////////////////////////////////////

#include <iostream>

#include "cuda_util.h"

////////////////////////////////////////////////////////////////////////////////////////////////////
// add constant to vector
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void
vecAddC(float* data, const unsigned int n, const float val) {

	data[threadIdx.x] += val;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// program entry point
////////////////////////////////////////////////////////////////////////////////////////////////////
int
main(int /*argc*/, char** /*argv*/) {

	const unsigned int n = 16;
	float* data = (float*)malloc(sizeof(float) * n);
	for (unsigned int i = 0; i < n; ++i) {
		data[i] = (float)i;
	}

	// get number of CUDA devices
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	std::cerr << deviceCount << std::endl;

	// query the device properties
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);
	printDeviceProps(props);

	// set the device
	hipSetDevice(0);

	// initialize memory
	float* device_data;

	// allocate device memory
	hipMalloc((void**)&device_data, sizeof(float) *n);

	// copy to device memory
	hipMemcpy(device_data, data, sizeof(float) * n, hipMemcpyHostToDevice);

	// determine thread layout


	// run kernel
	//make 16 threads and do vecAddC in all of them
	vecAddC << < 1, 16 >> >(device_data, n, 5);
	hipDeviceSynchronize();

	// copy result back to host
	hipMemcpy(data, device_data, sizeof(float) * n, hipMemcpyDeviceToHost);

	// output
	for (unsigned int i = 0; i < n; ++i) {
		std::cout << "data[" << i << "] = " << data[i] << '\n';
	}

	// clean up device memory
	hipFree(device_data);

	// clean up host memory
	free(data);

	std::getchar();
	return EXIT_SUCCESS;
}